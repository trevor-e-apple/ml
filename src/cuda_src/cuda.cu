
#include <hip/hip_runtime.h>
__declspec(dllexport) void init_cuda_device();
__declspec(dllexport) void *alloc_cuda_mem(size_t byte_count);
__declspec(dllexport) void *cuda_free();
__declspec(dllexport) void cuda_mem_copy_to_device(void *dst, void *src, size_t byte_count);
__declspec(dllexport) void cuda_mem_copy_to_host(void *dst, void *src, size_t byte_count);
__declspec(dllexport) void add(float *a, float *b, size_t len, float *out);

void init_cuda_device()
{
}

void *alloc_cuda_mem(size_t byte_count)
{
    void *result = NULL;
    hipMalloc(&result, byte_count);
    return result;
}

void cuda_free(void *mem)
{
    hipFree(mem);
}

void cuda_mem_copy_to_device(void *dst, void *src, size_t byte_count)
{
    hipMemcpy(dst, src, byte_count, hipMemcpyHostToDevice);
}

void cuda_mem_copy_to_host(void *dst, void *src, size_t byte_count)
{
    hipMemcpy(dst, src, byte_count, hipMemcpyDeviceToHost);
}

__global__ void add_kernel(float *a, float *b, size_t len, float *out)
{
    for (int i = 0; i < len; i++)
    {
        out[i] = a[i] + b[i];
    }
}

void add(float *a, float *b, size_t len, float *out)
{
    add_kernel<<<1, 1>>>(a, b, len, out);
}